﻿#include <iostream>
#include <ctime>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bitmap.h"
#include "image_processing.h"

int main() {
	Bitmap bmp("cat.bmp");

	ImageProcessing::MakeGreyScaleGPU(&bmp, MemoryTypeGPU::TEXTURE);
	bmp.GenerateBitmapImage("cat_greyscale.bmp");

	ImageProcessing::SobelOperatorGPU(&bmp, MemoryTypeGPU::GLOBAL
	);
	//ImageProcessing::SobelOperatorCPU(&bmp);
	bmp.GenerateBitmapImage("cat_sobel.bmp");

	return 0;
}