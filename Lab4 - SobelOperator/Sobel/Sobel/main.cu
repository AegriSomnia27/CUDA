﻿#include "hip/hip_runtime.h"
#include ""

#include "bitmap.h"


int main() {
	Bitmap bmp("cat.bmp", "test.bmp");
	bmp.DisplayImageInfo();
	bmp.GenerateBitmapImage();

	return 0;
}