﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <iostream>

#include "bitmap.h"


int main() {
	Bitmap bmp("cat.bmp", "testshit.bmp");
	bmp.GenerateBitmapImage();

	return 0;
}